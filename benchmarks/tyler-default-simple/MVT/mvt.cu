#include "hip/hip_runtime.h"
/**
 * mvt.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../common/UVMBench/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define N PSIZE

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;
size_t PSIZE;



void init_array(DATA_TYPE* A, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1, DATA_TYPE* y2)
{
    int i, j;

    #pragma omp parallel private(i, j) shared(A, x1, x2, y1, y2)
    {
        #pragma omp for simd nowait
        for (i = 0; i < N; i++)
        {
            x1[i] = ((DATA_TYPE) i) / N;
            x2[i] = ((DATA_TYPE) i + 1) / N;
            y1[i] = ((DATA_TYPE) i + 3) / N;
            y2[i] = ((DATA_TYPE) i + 4) / N;
        }

        #pragma omp for simd collapse(2)
        for (i = 0; i < N; i++)
        {
            for (j = 0; j < N; j++)
            {
                A[i * N + j] = ((DATA_TYPE) i * j) / N;
            }
        }
    }
}



void runMvt(DATA_TYPE* a, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1, DATA_TYPE* y2)
{
	int i, j;
	
	for (i=0; i<N; i++) 
	{
		for (j=0; j<N; j++) 
		{
       			x1[i] = x1[i] + a[i*N + j] * y1[j];
        	}
    	}
	
	for (i=0; i<N; i++) 
	{
		for (j=0; j<N; j++) 
		{
 		       	x2[i] = x2[i] + a[j*N + i] * y2[j];
      		}
    	}
}


void compareResults(DATA_TYPE* x1, DATA_TYPE* x1_outputFromGpu, DATA_TYPE* x2, DATA_TYPE* x2_outputFromGpu)
{
	int i, fail;
	fail = 0;
	
	for (i=0; i<N; i++) 
	{
		if (percentDiff(x1[i], x1_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}

		if (percentDiff(x2[i], x2_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void mvt_kernel1(DATA_TYPE *a, DATA_TYPE *x1, DATA_TYPE *y_1, size_t PSIZE)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		int j;
		for(j=0; j < N; j++)
		{
			x1[i] += a[i * N + j] * y_1[j];
		}
	}
}


__global__ void mvt_kernel2(DATA_TYPE *a, DATA_TYPE *x2, DATA_TYPE *y_2, size_t PSIZE)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		int j;
		for(j=0; j < N; j++)
		{
			x2[i] += a[j * N + i] * y_2[j];	
		}
	}
}

void mvtCuda(DATA_TYPE* a_gpu, DATA_TYPE* x1_gpu, DATA_TYPE* x2_gpu, DATA_TYPE* y_1_gpu, DATA_TYPE* y_2_gpu)
{
	double t_start, t_end;
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil((float)N/ ((float)DIM_THREAD_BLOCK_X)), 1);
	t_start = rtclock();
	mvt_kernel1<<<grid,block>>>(a_gpu,x1_gpu,y_1_gpu, PSIZE);
	mvt_kernel2<<<grid,block>>>(a_gpu,x2_gpu,y_2_gpu, PSIZE);
	hipDeviceSynchronize();
	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
}


int main(int argc, char * argv[])
{
	//Set problem size with argv[1]
        if(argc>=2){
                if(strcmp(argv[1],"-h")==0){
                        printf("Usage: %s <psize (GB)> [1: cpu, 0: no cpu]\n",argv[0]);
                        exit(0);
                }
                double bytes = 1024. * 1024. * 1024. * atof(argv[1]);
                //printf("Bytes: %.2f\n", bytes);
                PSIZE = (size_t) (sqrt(bytes/4));
        }
        else{
                PSIZE = 4096;
        }
        //printf("PSIZE: %zu\n", PSIZE);
        printf("Problem size: %.2f GB\n", ((((double)(PSIZE * PSIZE) + (4 * PSIZE)) * 4)/(1024. * 1024. * 1024.)));
        int cpu = 0;
        if(argc >= 3)
                cpu = atoi(argv[2]);	



	double t_start, t_end;

	DATA_TYPE* a = NULL;
	DATA_TYPE* x1 = NULL;
	DATA_TYPE* x2 = NULL;
	DATA_TYPE* y_1 = NULL;
	DATA_TYPE* y_2 = NULL;
	DATA_TYPE* a_gpu = NULL;
	DATA_TYPE* x1_gpu = NULL;
	DATA_TYPE* x2_gpu = NULL;
	DATA_TYPE* y_1_gpu = NULL;
	DATA_TYPE* y_2_gpu = NULL;
	hipMallocManaged(&a_gpu, sizeof(DATA_TYPE) * N * N);
	hipMallocManaged(&x1_gpu, sizeof(DATA_TYPE) * N);
	hipMallocManaged(&x2_gpu, sizeof(DATA_TYPE) * N);
	hipMallocManaged(&y_1_gpu, sizeof(DATA_TYPE) * N);
	hipMallocManaged(&y_2_gpu, sizeof(DATA_TYPE) * N);
	
	printf("Start address of a_gpu:\t%p\n", &(a_gpu[0]));
	printf("Start address of x1_gpu:\t%p\n", &(x1_gpu[0]));
	printf("Start address of x2_gpu:\t%p\n", &(x2_gpu[0]));
	printf("Start address of y_1_gpu:\t%p\n", &(y_1_gpu[0]));
	printf("Start address of y_2_gpu:\t%p\n", &(y_2_gpu[0]));
	
	init_array(a_gpu, x1_gpu, x2_gpu, y_1_gpu, y_2_gpu);
	
	GPU_argv_init();

	mvtCuda(a_gpu, x1_gpu, x2_gpu, y_1_gpu, y_2_gpu);
	
	if(cpu)
    {
        a = (DATA_TYPE*)malloc(N*N*sizeof(DATA_TYPE));
        x1 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
        x2 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
        y_1 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
        y_2 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));

	    init_array(a, x1, x2, y_1, y_2);
		
        t_start = rtclock();
		//run the algorithm on the CPU
		runMvt(a, x1, x2, y_1, y_2);

		t_end = rtclock();
		fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
		
		compareResults(x1, x1_gpu, x2, x2_gpu);
        free(a);
        free(x1);
        free(x2);
        free(y_1);
        free(y_2);
	}
	hipFree(a_gpu);
	hipFree(x1_gpu);
	hipFree(x2_gpu);
	hipFree(y_1_gpu);
	hipFree(y_2_gpu);
  	return 0;
}

