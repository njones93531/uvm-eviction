#include "hip/hip_runtime.h"
/**
 * fdtd2d.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "accpol.h"
#include "../common/UVMBench/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 10.05

#define GPU_DEVICE 0

/* Problem size */
#define tmax 10
#define NX PSIZE
#define NY PSIZE

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;
size_t PSIZE;


void init_arrays(DATA_TYPE* hz_gpu)
{
	int i, j;

  #pragma omp parallel for
	for (i = 0; i < NX; i++)
	{
		for (j = 0; j < NY; j++)
		{
			hz_gpu[i*NY + j] = ((DATA_TYPE) (i-9)*(j+4) + 3) / NX;
		}
	}
}


void runFdtd(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
	int t, i, j;
	
	for (t=0; t < tmax; t++)  
	{
		for (j=0; j < NY; j++)
		{
			ey[0*NY + j] = _fict_[t];
		}
	
		for (i = 1; i < NX; i++)
		{
       		for (j = 0; j < NY; j++)
			{
       			ey[i*NY + j] = ey[i*NY + j] - 0.5*(hz[i*NY + j] - hz[(i-1)*NY + j]);
        		}
		}

		for (i = 0; i < NX; i++)
		{
       		for (j = 1; j < NY; j++)
			{
				ex[i*(NY+1) + j] = ex[i*(NY+1) + j] - 0.5*(hz[i*NY + j] - hz[i*NY + (j-1)]);
			}
		}

		for (i = 0; i < NX; i++)
		{
			for (j = 0; j < NY; j++)
			{
				hz[i*NY + j] = hz[i*NY + j] - 0.7*(ex[i*(NY+1) + (j+1)] - ex[i*(NY+1) + j] + ey[(i+1)*NY + j] - ey[i*NY + j]);
			}
		}
	}
}

__global__ void compareResultsKernel(DATA_TYPE* hz1, DATA_TYPE* hz2, int* fail, int PSIZE) {
    // Shared memory for partial sum within a block
    __shared__ int block_fail_count[256];  // assuming up to 256 threads per block

    // Get thread indexes
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Row index
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Column index

    // Initialize shared memory
    int thread_fail = 0;
    if (i < NX && j < NY) {
        if (percentDiff(hz1[i * NY + j], hz2[i * NY + j]) > PERCENT_DIFF_ERROR_THRESHOLD) {
            thread_fail = 1;
        }
    }

    // Perform reduction within a block
    int thread_id = threadIdx.x + blockDim.x * threadIdx.y;  // unique thread ID within block
    block_fail_count[thread_id] = thread_fail;
    __syncthreads();

    // Reduction: sum of block_fail_count
    for (int stride = blockDim.x * blockDim.y / 2; stride > 0; stride /= 2) {
        if (thread_id < stride) {
            block_fail_count[thread_id] += block_fail_count[thread_id + stride];
        }
        __syncthreads();
    }

    // Accumulate the result from this block to global memory (only by one thread)
    if (thread_id == 0) {
        atomicAdd(fail, block_fail_count[0]);
    }
}

void compareResultsCUDA(DATA_TYPE* hz1, DATA_TYPE* hz2) {
    int fail = 0;
    int *d_fail;

    // Allocate memory for fail counter on the device
    hipMalloc((void **)&d_fail, sizeof(int));
    hipMemcpy(d_fail, &fail, sizeof(int), hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 block(256, 256);  // 16x16 threads per block
    dim3 grid((NX + block.x - 1) / block.x, (NY + block.y - 1) / block.y);  // Ensure coverage of all elements

    // Launch the kernel
    compareResultsKernel<<<grid, block>>>(hz1, hz2, d_fail, PSIZE);

    // Copy the fail result back to host
    hipMemcpy(&fail, d_fail, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_fail);

    // Print results
    printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void compareResults(DATA_TYPE* hz1, DATA_TYPE* hz2)
{
	int i, j, fail;
	fail = 0;
	
	for (i=0; i < NX; i++) 
	{
		for (j=0; j < NY; j++) 
		{
			if (percentDiff(hz1[i*NY + j], hz2[i*NY + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}

__global__ void fdtd_stream_kernel(DATA_TYPE* hz, size_t PSIZE)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NX) && (j < NY))
	{
		hz[j * NY + i]++;
  }
 
}
__global__ void fdtd_step1_kernel(DATA_TYPE *hz, int t, size_t PSIZE)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NX) && (j < NY))
	{
		if (i > 0)
		{ 
			 if( 0.5f*(hz[i * NY + j] - hz[(i-1) * NY + j]) == -123123123)
       {
         hz[i * NY + j] = 1;
       }
		}
	}
}



__global__ void fdtd_step2_kernel(DATA_TYPE *hz, int t, size_t PSIZE)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < NX) && (j < NY) && (j > 0))
	{
		if( 0.5f*(hz[i * NY + j] - hz[i * NY + (j-1)]) == -123123123)
    {
      hz[i * NY + j]++;
    }
	}
}


__global__ void fdtd_step3_kernel(DATA_TYPE *hz, int t, size_t PSIZE)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < NX) && (j < NY))
	{	
		hz[i * NY + j] = hz[i * NY + j] - 0.7f;
	}
}


// Function to round up to the next power of 2
__host__ __device__
size_t nextPowerOf2(size_t n) {
    if (n == 0) return 1; // Special case for 0
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n |= n >> 32;  // This works for 64-bit integers
    return n + 1;
}


void fdtdCuda(DATA_TYPE* hz_gpu)
{
	double t_start, t_end, t_total;

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid( nextPowerOf2((size_t)ceil(((float)NY) / ((float)block.x))), nextPowerOf2((size_t)ceil(((float)NX) / ((float)block.y))));


  //Ensure that the pages of hz_gpu are arranged IN ORDER on the device
  //hipMemPrefetchAsync(hz_gpu, PSIZE*PSIZE * sizeof(DATA_TYPE), 0 /* device ID */, nullptr);
  //hipDeviceSynchronize();
  //fdtd_stream_kernel<<<grid,block>>>(hz_gpu, PSIZE);
  //hipDeviceSynchronize();

	t_total = rtclock();
	for(int t = 0; t< tmax; t++)
	{
	  t_start = rtclock();
		fdtd_step1_kernel<<<grid,block>>>(hz_gpu, t, PSIZE);
		hipDeviceSynchronize();
	  t_end = rtclock();
    fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	  t_start = rtclock();
		fdtd_step2_kernel<<<grid,block>>>(hz_gpu, t, PSIZE);
		hipDeviceSynchronize();
	  t_end = rtclock();
    fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	  t_start = rtclock();
		fdtd_step3_kernel<<<grid,block>>>(hz_gpu, t, PSIZE);
		hipDeviceSynchronize();
	  t_end = rtclock();
    fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	  t_start = rtclock();
	}
  t_total = rtclock() - t_total;
  fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_total);
	
}


int main(int argc, char * argv[])
{

	//Set problem size with argv[1]
        if(argc>=2){
                if(strcmp(argv[1],"-h")==0){
                        printf("Usage: %s <psize (GB)> [1: cpu, 0: no cpu]\n",argv[0]);
                        exit(0);
                }
                double bytes = 1024. * 1024. * 1024. * atof(argv[1]);
                //printf("Bytes: %.2f\n", bytes);
                PSIZE = (size_t) (sqrt((bytes-tmax)/12));
        }
        else{
                PSIZE = 2048;
        }
        //printf("PSIZE: %zu\n", PSIZE)        printf("Problem size: %.2f GB\n", ((((double)(PSIZE * PSIZE * 3) + (TMAX)) * 4)/(1024. * 1024. * 1024.)));
        printf("Problem size: %.2f GB\n", ((((double)(PSIZE * PSIZE * 3) + (tmax)) * 4)/(1024. * 1024. * 1024.)));
        int cpu = 0;
	if(argc >= 3)
                cpu = atoi(argv[2]);	



	double t_start, t_end;

	DATA_TYPE *hz_gpu;

//	_fict_ = (DATA_TYPE*)malloc(tmax*sizeof(DATA_TYPE));
//	ex = (DATA_TYPE*)malloc(NX*(NY+1)*sizeof(DATA_TYPE));
//	ey = (DATA_TYPE*)malloc((NX+1)*NY*sizeof(DATA_TYPE));
	//hz = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));

	//hipMallocManaged(&_fict_gpu, sizeof(DATA_TYPE) * tmax);
	//hipMallocManaged(&ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1));
	//hipMallocManaged(&ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY);
	hipMallocManaged(&hz_gpu, sizeof(DATA_TYPE) * NX * NY);
	//hipMallocManaged(&hz, sizeof(DATA_TYPE) * NX * NY);

	init_arrays(hz_gpu);

	GPU_argv_init();
	
	AccessPolicy acp;
	//acp.setAllocationPolicy((void**)&_fict_gpu, sizeof(DATA_TYPE) * tmax, 0, argc, argv);
	//Set fict_gpu to d
  size_t size = sizeof(DATA_TYPE) * tmax;
 // hipMalloc(&devptr, size);
 // CHECK_CUDA_ERROR();
 // hipMemcpy(devptr, *a, size, hipMemcpyHostToDevice);
 // CHECK_CUDA_ERROR();
 // CUDA_CHECK(hipFree(*a));
 // CHECK_CUDA_ERROR();
 // *a = devptr;

 // acp.setAllocationPolicy((void**)&ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1), 0, argc, argv);
	//acp.setAllocationPolicy((void**)&ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY, 1, argc, argv);
	acp.setAllocationPolicy((void**)&hz_gpu, sizeof(DATA_TYPE) * NX * NY, 0, argc, argv);

	
	fdtdCuda(hz_gpu);

	acp.freeMemPressure();

	hipFree(hz_gpu);
	return 0;
}

